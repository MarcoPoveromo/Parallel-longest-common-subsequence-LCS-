#include "hip/hip_runtime.h"
//
// Created by marco on 18/07/22.
//

#include "LCS_v2.cuh"
#include "../../cuda_launch_config.cu"

// --------------------------------- BEGIN CUDA ------------------------------------------------
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__
int mapper(char x){
    if (x == 'A') return 0;
    if (x == 'C') return 1;
    if (x == 'G') return 2;
    return 3;
}

__global__
void computeLcs_v2(int *precRow, int *currentRow, int *P, int ySize, char *X, int i, char x_i) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j == 0) {
        currentRow[j] = 0;
        return;
    }

    bool t, s;
    int upper, PMapper;

    while(j <= ySize) {
        upper = precRow[j];
        PMapper = P[mapper(x_i) * (ySize+1) + j];

        t = (0 - PMapper) < 0;
        s = (0 - (upper - t * precRow[PMapper - 1])) < 0;
        currentRow[j] = precRow[j] + t * !s;

        j += blockDim.x * gridDim.x;
    }
}

// ---------------------------------- END CUDA -------------------------------------------------

LCS_v2::LCS_v2(const std::string &x, const std::string &y, const dim3 &nBlocks, const dim3 &nThreadsPerBlock)
        : LCSStrategy(x, y), nBlocks(nBlocks), nThreadsPerBlock(nThreadsPerBlock) {}

LCS_v2::LCS_v2(const std::string &x, const std::string &y) : LCSStrategy(x, y) {}

const dim3 &LCS_v2::getNBlocks() const {
    return nBlocks;
}

void LCS_v2::setNBlocks(const dim3 &nBlocks) {
    this->nBlocks = nBlocks;
}

const dim3 &LCS_v2::getNThreadsPerBlock() const {
    return nThreadsPerBlock;
}

void LCS_v2::setNThreadsPerBlock(const dim3 &nThreadsPerBlock) {
    this->nThreadsPerBlock = nThreadsPerBlock;
}

Result LCS_v2::computeLCS() {
    // Start timer event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    size_t size = sizeof(int) * (Y.size() + 1);
    int *d_currentRow, *d_precRow;
    hipMalloc(&d_precRow, size);
    hipMalloc(&d_currentRow, size);
    hipMemset(d_currentRow, 0, size);
    // Copy X and Y on GPU
    char *d_X;
    hipMalloc(&d_X, X.size() + 1);
    hipMemcpy(d_X, X.c_str(), sizeof(char) * (X.size() + 1), hipMemcpyHostToDevice);
    int l = 4;
    int C[] = {'A', 'C', 'G', 'T'};

    // Initialize P lookups table
    size_t size_P = sizeof(int) * l * (Y.size() + 1);
    int *P = (int *) malloc(size_P);
    int *d_P;
    hipMalloc(&d_P, size_P);

    for (int i = 0; i < l; i++) {
        for (int j = 0; j <= Y.size(); j++) {
            if(j == 0) P[i * (Y.size()+1) + j] = 0;
            else if (Y[j-1] == C[i]) P[i * (Y.size()+1) + j] = j;
            else P[i * (Y.size()+1) + j] = P[i * (Y.size()+1) + j - 1];
        }
    }

    hipMemcpy(d_P, P, size_P, hipMemcpyHostToDevice);

    // Maximize occupancy https://github.com/jaredhoberock/cuda_launch_config
    size_t num_threads = block_size_with_maximum_potential_occupancy(computeLcs_v2);
    size_t num_blocks = (Y.size()+1) / num_threads;
    if((Y.size()+1) % num_threads) ++num_blocks;

    for(int i = 1; i <= X.size(); i++) {
        char x_i = X[i-1];
        if(i%2==0) computeLcs_v2<<<num_blocks, num_threads>>>(d_precRow, d_currentRow, d_P, Y.size(), d_X, i, x_i);
        else computeLcs_v2<<<num_blocks, num_threads>>>(d_currentRow, d_precRow, d_P, Y.size(), d_X, i, x_i);
    }

    int *lcs = (int *) malloc(sizeof(int));
    if((X.size()) %2==0) hipMemcpy(lcs, &(d_currentRow[Y.size()]), sizeof(int), hipMemcpyDeviceToHost);
    else hipMemcpy(lcs, &(d_precRow[Y.size()]), sizeof(int), hipMemcpyDeviceToHost);


    // Stop timer event
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Build the result object
    Result result(std::to_string(*lcs));
    result.setTimeElapsed(milliseconds);

    hipFree(d_precRow); hipFree(d_currentRow); hipFree(d_X);

    return result;
}
