#include "hip/hip_runtime.h"
//
// Created by marco on 18/07/22.
//

#include "LCS_v1.cuh"

#include "../../cuda_launch_config.cu"

// --------------------------------- BEGIN CUDA ------------------------------------------------
/* Old kernel that use the entire matrix
__global__
void lcs(int *M, int i, int j, char *X, char *Y, int ySize, int diagonalSize) {
    int d = blockIdx.x * blockDim.x + threadIdx.x;
    int a, b;
    while (d <= diagonalSize) {

        a = i + d;
        b = j - d;

        if (a == 0 || b == 0) M[a * (ySize + 1) + b] = 0;
        else if (X[a - 1] == Y[b - 1]) M[a * (ySize + 1) + b] = M[(a - 1) * (ySize + 1) + b - 1] + 1;
        else M[a * (ySize + 1) + b] = max(M[(a - 1) * (ySize + 1) + b], M[a * (ySize + 1) + b - 1]);

        d += blockDim.x * gridDim.x;
    }
}*/

__global__
void lcs(int *precPrecDiagonal, int *precDiagonal, int *currentDiagonal, int i, int j, char *X, char *Y, int diagonalSize) {
    int d = blockIdx.x * blockDim.x + threadIdx.x;

    int a, b;

    while (d <= diagonalSize) {
        a = i + d;
        b = j - d;

        if (a == 0 || b == 0) currentDiagonal[b] = 0;
        else if (X[a - 1] == Y[b - 1]) currentDiagonal[b] = precPrecDiagonal[b - 1] + 1;
        else currentDiagonal[b] = max(precDiagonal[b], precDiagonal[b - 1]);

        d += blockDim.x * gridDim.x;
    }
}
// ---------------------------------- END CUDA -------------------------------------------------

LCS_v1::LCS_v1(const std::string &x, const std::string &y, const dim3 &nBlocks, const dim3 &nThreadsPerBlock)
        : LCSStrategy(x, y), nBlocks(nBlocks), nThreadsPerBlock(nThreadsPerBlock) {}

LCS_v1::LCS_v1(const std::string &x, const std::string &y) : LCSStrategy(x, y) {}

const dim3 &LCS_v1::getNBlocks() const {
    return nBlocks;
}

void LCS_v1::setNBlocks(const dim3 &nBlocks) {
    this->nBlocks = nBlocks;
}

const dim3 &LCS_v1::getNThreadsPerBlock() const {
    return nThreadsPerBlock;
}

void LCS_v1::setNThreadsPerBlock(const dim3 &nThreadsPerBlock) {
    this->nThreadsPerBlock = nThreadsPerBlock;
}

/* Old compute LCS
Result LCS_v1::computeLCS() {
    // Allocate memory for the 2D matrix in GPU
    int *d_M;
    size_t size = sizeof(int) * (X.size() + 1) * (Y.size() + 1);
    hipMalloc(&d_M, size);

    // Copy X and Y on GPU
    char *d_X, *d_Y;
    hipMalloc(&d_X, X.size() + 1);
    hipMalloc(&d_Y, Y.size() + 1);
    hipMemcpy(d_X, X.c_str(), sizeof(char) * (X.size() + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y.c_str(), sizeof(char) * (Y.size() + 1), hipMemcpyHostToDevice);

    int m = Y.size();
    int n = X.size();
    int diagonalSize;

    for (int i=0, j=0; i <= m && j <= n; j++) {
        diagonalSize = std::min(j, m - i);
        lcs<<<20, 512>>>(d_M, i, j, d_X, d_Y, Y.size(), diagonalSize);
        if(j == n){ j--; i++; }
    }
    hipDeviceSynchronize();


    int *M = (int *) malloc(size);
    hipMemcpy(M, d_M, size, hipMemcpyDeviceToHost);

    // Put the result in string for the moment
    std::string s = std::to_string(M[(X.size()+1) * (Y.size()+1) - 1]);

    // Free memory
    free(M); hipFree(d_M); hipFree(d_X); hipFree(d_Y);

    return Result(s);
}*/

Result LCS_v1::computeLCS() {
    // Start timer event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Allocate memory for the 2D matrix in GPU
    int *d_precPrecDiagonal, *d_precDiagonal, *d_currentDiagonal;
    size_t size = sizeof(int) * (Y.size() + 1);
    hipMalloc(&d_precPrecDiagonal, size);
    hipMalloc(&d_precDiagonal, size);
    hipMalloc(&d_currentDiagonal, size);

    // Copy X and Y on GPU
    char *d_X, *d_Y;
    hipMalloc(&d_X, X.size() + 1);
    hipMalloc(&d_Y, Y.size() + 1);
    hipMemcpy(d_X, X.c_str(), sizeof(char) * (X.size() + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y.c_str(), sizeof(char) * (Y.size() + 1), hipMemcpyHostToDevice);

    int m = Y.size();
    int n = X.size();
    int diagonalSize;

    size_t num_threads = block_size_with_maximum_potential_occupancy(lcs);

    for (int i=0, j=0; i <= m && j <= n; j++) {
        diagonalSize = std::min(j, m - i);

        lcs<<<ceil((diagonalSize+1) / (double)num_threads), num_threads>>>(d_precPrecDiagonal, d_precDiagonal, d_currentDiagonal, i, j, d_X, d_Y, diagonalSize);
        if(j == n){ j--; i++; }

        int *temp = d_precDiagonal;
        d_precDiagonal = d_currentDiagonal;
        int *temp2 = d_precPrecDiagonal;
        d_precPrecDiagonal = temp;
        d_currentDiagonal = temp2;
    }

    // Retrive lcs from GPU
    int *lcs = (int *) malloc(sizeof(int));
    hipMemcpy(lcs, &(d_precDiagonal[n]), sizeof(int), hipMemcpyDeviceToHost);

    // Stop timer event
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Build the result object
    Result result(std::to_string(*lcs));
    result.setTimeElapsed(milliseconds);

    // Free memory
    hipFree(d_precDiagonal);
    hipFree(d_currentDiagonal);
    hipFree(d_Y);
    hipFree(d_X);

    return result;
}